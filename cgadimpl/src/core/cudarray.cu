#include <hip/hip_runtime.h>
#include <iostream>





__global__ void adding_cuda(const float* A, const float* B, float* C, int width)
{
   int bx = blockIdx.x;
    int tx = threadIdx.x;

    int row = bx * blockDim.x + tx;

    float acc = 0.0f;



    // Accumulate into existing C value instead of overwriting
    if(row<width)
                C[row] = A[row] + B[row];
}














void run_cuda_add(const float* A, const float* B, float* C, int width)
{
    int threads = 256;
    int blocks = (width + threads - 1) / threads;

    adding_cuda<<<blocks, threads>>>(A, B, C, width);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: "
                  << hipGetErrorString(err) << std::endl;
    }

    hipDeviceSynchronize();
}



// =====================================================
// Elementwise Subtraction
// =====================================================
__global__ void subbing_cuda(const float* A, const float* B, float* C, int width)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < width)
        C[idx] = A[idx] - B[idx];
}

 void run_cuda_sub(const float* A, const float* B, float* C, int width)
{
    int threads = 256;
    int blocks = (width + threads - 1) / threads;

    subbing_cuda<<<blocks, threads>>>(A, B, C, width);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cerr << "CUDA SUB kernel launch error: "
                  << hipGetErrorString(err) << std::endl;

    hipDeviceSynchronize();
}


// =====================================================
// Elementwise Division
// =====================================================
__global__ void diving_cuda(const float* A, const float* B, float* C, int width)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < width)
        C[idx] = B[idx] != 0.0f ? A[idx] / B[idx] : 0.0f; // safe divide
}

 void run_cuda_div(const float* A, const float* B, float* C, int width)
{
    int threads = 256;
    int blocks = (width + threads - 1) / threads;

    diving_cuda<<<blocks, threads>>>(A, B, C, width);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cerr << "CUDA DIV kernel launch error: "
                  << hipGetErrorString(err) << std::endl;

    hipDeviceSynchronize();
}


// =====================================================
// Elementwise Multiplication (Hadamard Product)
// =====================================================
__global__ void muling_cuda(const float* A, const float* B, float* C, int width)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < width)
        C[idx] = A[idx] * B[idx];
}

 void run_cuda_hadmul(const float* A, const float* B, float* C, int width)
{
    int threads = 256;
    int blocks = (width + threads - 1) / threads;

    muling_cuda<<<blocks, threads>>>(A, B, C, width);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cerr << "CUDA HADMUL kernel launch error: "
                  << hipGetErrorString(err) << std::endl;

    hipDeviceSynchronize();
}
